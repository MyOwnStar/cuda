#include "hip/hip_runtime.h"
#include "kernel.h"

#include <hip/hip_runtime.h>
#include <>

#include <cmath>

#define TILE_SIZE 16


// ЗАКОММЕНТИРОВАЛ в ./crt/gost_runtime.h 243 строчку, заработало

__global__ void multMatrixGPU(const float *matrixA, const size_t rowA, const size_t columnA,
                           const float *matrixB, const size_t rowB, const size_t columnB, float *matrixC)
{
   __shared__ float ds_A[TILE_SIZE][TILE_SIZE];
   __shared__ float ds_B[TILE_SIZE][TILE_SIZE];

   int tx = threadIdx.x;
   int ty = threadIdx.y;

   int row = blockIdx.y * blockDim.y + ty;
   int col = blockIdx.x * blockDim.x + tx;

   float cValue = 0.0;

   for (int t = 0; t < (columnA - 1) / TILE_SIZE + 1; t++)
   {
      if (t * TILE_SIZE + tx < columnA && row < rowA)
      {
         ds_A[ty][tx] = matrixA[row * columnA + t * TILE_SIZE + tx];
      } else {
         ds_A[ty][tx] = 0.0;
      }

      if (t * TILE_SIZE + ty < rowB && col < columnB)
      {
         ds_B[ty][tx] = matrixB[(t * TILE_SIZE + ty) * columnB + col];
      } else {
         ds_B[ty][tx] = 0.0;
      }

      __syncthreads();

      for (int i = 0; i < TILE_SIZE; i++) {
         cValue += ds_A[ty][i] * ds_B[i][tx];
      }

      __syncthreads();
   }

   if (row < rowA && col < columnB) {
      matrixC[row * columnB + col] = cValue;
   }
}

void multMatrix(const float *matrixA, const size_t rowA, const size_t columnA,
                const float *matrixB, const size_t rowB, const size_t columnB, float *matrixC)
{
   dim3 threadsPerBlock(TILE_SIZE, TILE_SIZE, 1);
   dim3 blocksPerGrid((columnB - 1) / TILE_SIZE + 1, (rowA -1) / TILE_SIZE + 1, 1);

//   int numBlock;
//   int blockSize = 16;
//
//   hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlock, multMatrixGPU, blockSize, 0);
//
//   int gridSize = (N * N + blockSize - 1) / blockSize;

   //std::cout << "BlockPerGrid: " << blocksPerGrid.x << "x" << blocksPerGrid.y << "\n";
   //std::cout << "ThreadsPerBlock: " << threadsPerBlock.y << "x" << threadsPerBlock.y << "\n\n";

   multMatrixGPU<<<blocksPerGrid, threadsPerBlock>>>(matrixA, rowA, columnB,
                                                     matrixB, rowB, columnB, matrixC);
   //multMatrix <<<gridSize, blockSize >>>(a, b, c, N);
}