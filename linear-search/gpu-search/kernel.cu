#include "hip/hip_runtime.h"
#include "kernel.h"

#include <hip/hip_runtime.h>
#include <>
#include <cmath>


__global__ void searchGPU(float *data, const float x, int *result)
{
   int idx = blockIdx.x * blockDim.x + threadIdx.x;

   if (data[idx] == x)
   {
      result[0] = static_cast<int>(data[idx]);
      result[1] = idx;
   }
}

void linearSearchGPU(float *data, const float x, int *result, const size_t n)
{
   int threads = 512;
   int blocks = n / threads;
//   dim3 threads(n, n);
//   dim3 blocks(1, 1);
//
//   if (n*n > 512){
//      threads.x = 512;
//      threads.y = 512;
//      blocks.x = ceil(double(n)/double(threads.x));
//      blocks.y = ceil(double(n)/double(threads.y));
//   }

   searchGPU<<< blocks, threads>>>(data, x, result);
}