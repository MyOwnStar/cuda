#include "hip/hip_runtime.h"
#include "kernel.h"

#include <hip/hip_runtime.h>
#include <>


__device__ float f(float x)
{
   return 4.f / (1.f + x * x);
}

__global__ void calcIntegralGPU(float *array, const float h, const long stepCount,
                                                             const int threads,
                                                             const int blocks)
{
   int idx = blockIdx.x * blockDim.x + threadIdx.x;

   for (int i = idx; i < stepCount; i+= threads * blocks)
   {
      float x = (i + 0.5f) * h;
      array[idx] += f(x);
   }
}

void calcIntegral(float *array, const float h, const long stepCount,
                                               const int threads,
                                               const int blocks)
{
   dim3 blocksPerGrid = 30;
   dim3 threadPerBlocks = 8;

   calcIntegralGPU<<< blocksPerGrid, threadPerBlocks >>>(array, h, stepCount, threads, blocks);
}