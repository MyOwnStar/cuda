#include "hip/hip_runtime.h"
#include "kernel.h"

#include <hip/hip_runtime.h>
#include <>

#define BLOCK_SIZE 16

__global__ void transGPU(const float *inMatrix, float *outMatrix, const size_t row, const size_t column)
{
    size_t xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    size_t yIndex = blockIdx.y * blockDim.y + threadIdx.y;

    if ((xIndex < column) && (yIndex < row))
    {
        size_t inIndex = yIndex * column + xIndex;
        size_t outIndex = xIndex * row + yIndex;

        outMatrix[outIndex] = inMatrix[inIndex];
    }
}

void transpose(const float *inputMatrix, float *outputMatrix, const size_t row, const size_t column)
{
    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE, 1);
    dim3 blocksPerGrid((column + BLOCK_SIZE - 1) / BLOCK_SIZE, (row + BLOCK_SIZE - 1) / BLOCK_SIZE, 1);

    transGPU<<< blocksPerGrid, threadsPerBlock >>>(inputMatrix, outputMatrix, row, column);
}
